#include <stdio.h>
#include <hip/hip_runtime.h>
#include "memory.h"
namespace lyramilk { namespace tensor {



	//	float* gf;
	memory<float>::memory()
	{
		ptr = nullptr;
		count = 0;
	}

	memory<float>::memory(const memory& ov)
	{
		ptr = nullptr;
		count = 0;
	}

	memory<float>::~memory()
	{
		if(ptr!=nullptr){
			hipFree(ptr);
		}
	}

	void memory<float>::assign(const float* p,long size)
	{
		count = size;
		hipMalloc((void**)&ptr, sizeof(float) * count);
		hipMemcpy(ptr, p, sizeof(float) * count, hipMemcpyHostToDevice);

	}

	void memory<float>::assign(const std::vector<float>& ov)
	{
		count = ov.size();
		hipMalloc((void**)&ptr, sizeof(float) * count);
		hipMemcpy(ptr, ov.data(), sizeof(float) * count, hipMemcpyHostToDevice);
	}

	memory<float>::operator std::vector<float>&&()
	{
		
	}

	long memory<float>::size()
	{
		return count;
	}



}}
